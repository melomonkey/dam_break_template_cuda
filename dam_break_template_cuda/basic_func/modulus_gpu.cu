#include "hip/hip_runtime.h"


__device__ double modulus_3_d_gpu(double * v)
{
	double tmp = v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
	tmp = pow(tmp, .5);
	return tmp;
}

__device__ float modulus_3_f_gpu(float * v)
{
	float tmp = v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
	tmp = powf(tmp, .5);
	return tmp;
}

__device__ int modulus_3_i_gpu(int * v)
{
	float tmp = v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
	tmp = powf(tmp, .5);
	return tmp;
}