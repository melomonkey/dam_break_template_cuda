#include "hip/hip_runtime.h"


__device__ double constrain_d_gpu(double val, double min, double max)
{
	double tmp = val;

	if (tmp <= min) tmp = min; if (tmp >= max) tmp = max;

	return tmp;
}

__device__ float constrain_f_gpu(float val, float min, float max)
{
	float tmp = val;

	if (tmp <= min) tmp = min; if (tmp >= max) tmp = max;

	return tmp;
}

__device__ int constrain_i_gpu(int val, int min, int max)
{
	int tmp = val;

	if (tmp <= min) tmp = min; if (tmp >= max) tmp = max;

	return tmp;
}