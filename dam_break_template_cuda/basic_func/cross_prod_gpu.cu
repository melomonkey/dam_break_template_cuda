
#include "hip/hip_runtime.h"


__device__ void cross_prod_3_d_gpu(double * v1, double * v2, double * ans)
{
	ans[0] = v1[1] * v2[2] - v1[2] * v2[1];
	ans[1] = v1[0] * v2[2] - v1[2] * v2[0];
	ans[2] = v1[0] * v2[1] - v1[1] * v2[0];
}

__device__ void cross_prod_3_f_gpu(float * v1, float * v2, float * ans)
{
	ans[0] = v1[1] * v2[2] - v1[2] * v2[1];
	ans[1] = v1[0] * v2[2] - v1[2] * v2[0];
	ans[2] = v1[0] * v2[1] - v1[1] * v2[0];
}

__device__ void cross_prod_3_i_gpu(int * v1, int * v2, int * ans)
{
	ans[0] = v1[1] * v2[2] - v1[2] * v2[1];
	ans[1] = v1[0] * v2[2] - v1[2] * v2[0];
	ans[2] = v1[0] * v2[1] - v1[1] * v2[0];
}