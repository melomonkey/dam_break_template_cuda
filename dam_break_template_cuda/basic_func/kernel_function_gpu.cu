#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "..\configuration.h"

__device__ double kernel_function_gpu(double dis, double h)
{
	double R = dis / h;
	double alpha;

	if (CASE_DIM == 0) alpha = (1.0 / (h * h)) * pow(PI, -1.0);
	if (CASE_DIM == 1) alpha = (1.0 / (h * h * h)) * pow(PI, -1.5);

	double ans = alpha * exp(-(R * R));
	return ans;
}

__device__ double kernel_function_1dev_gpu(double dis, double h)
{
	double R = dis / h;
	double alpha;

	if (CASE_DIM == 0) alpha = (1.0 / (h * h)) * pow(3.14, -1.0);
	if (CASE_DIM == 1) alpha = (1.0 / (h * h * h)) * pow(3.14, -1.5);

	double ans = alpha * exp(-(R * R)) * (-2.0 * R);

	return ans / h;
}

__device__ double kernel_function_2dev_gpu(double dis, double h)
{
	double R = dis / h;
	double alpha;

	if (CASE_DIM == 0) alpha = (1.0 / (h * h)) * pow(PI, -1.0);
	if (CASE_DIM == 1) alpha = (1.0 / (h * h * h)) * pow(PI, -1.5);

	double ans = -2.0 * alpha * (1 - 2 * R * R) * exp(-(R * R));

	return ans / h / h; // local particle's position coordination's derivative, so include the R's derivative, twice.
}