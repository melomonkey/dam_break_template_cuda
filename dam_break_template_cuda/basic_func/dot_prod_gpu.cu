#include "hip/hip_runtime.h"


__device__ double dot_prod_3_d_gpu(double * v1, double * v2)
{
	double tmp = 0.0;
	for (int i = 0; i < 3; ++i) tmp += v1[i] * v2[i];
	return tmp;
}

__device__ float dot_prod_3_f_gpu(float * v1, float * v2)
{
	float tmp = 0.0;
	for (int i = 0; i < 3; ++i) tmp += v1[i] * v2[i];
	return tmp;
}

__device__ int dot_prod_3_i_gpu(int * v1, int * v2)
{
	int tmp = 0.0;
	for (int i = 0; i < 3; ++i) tmp += v1[i] * v2[i];
	return tmp;
}