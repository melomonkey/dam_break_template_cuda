#include "hip/hip_runtime.h"
#include "..\struct_particle.h"
#include "..\configuration.h"
#include "..\basic_func\basic_func.h"
#include <vector>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

__global__ void _acc_pressure_term_update_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return; 

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	double local_x, local_y, local_z;
	local_x = particles[calcu_particle].coorX.val[0];
	local_y = particles[calcu_particle].coorY.val[0];
	local_z = particles[calcu_particle].coorZ.val[0];
	double h = particles[calcu_particle].smthR.val[0];

	double acc_x = 0.0,
		acc_y = 0.0,
		acc_z = 0.0;

	double _acc[3];
	for (int m = 0; m < 3; ++m) _acc[m] = 0.0;

	for (int j = 1; j < neigh_list_width; ++j)
	{
		if (neighbors_list[j + pos * neigh_list_width] == 0) break;

		unsigned int label_ij = neighbors_list[j + pos * neigh_list_width] - 1;

		if (particles[label_ij].id == 0 || particles[label_ij].id == 1 || particles[label_ij].id == 2 || particles[label_ij].id == 3) continue;

		double tmp_x, tmp_y, tmp_z;
		tmp_x = particles[label_ij].coorX.val[0];
		tmp_y = particles[label_ij].coorY.val[0];
		tmp_z = particles[label_ij].coorZ.val[0];

		double dis_x, dis_y, dis_z;
		dis_x = local_x - tmp_x;
		dis_y = local_y - tmp_y;
		dis_z = local_z - tmp_z;

		// Interactive particles
		if (particles[label_ij].id >= 4)
		{
			double coeff0;
			coeff0 = -(particles[calcu_particle].pressure.val[0] + particles[label_ij].pressure.val[0]) / (particles[calcu_particle].density.val[0] * particles[label_ij].density.val[0]);
			coeff0 *= particles[label_ij].mass.val[0];
			double de_kernel_val[3] = { kernel_function_1dev_gpu(dis_x, h), kernel_function_1dev_gpu(dis_y, h), kernel_function_1dev_gpu(dis_z, h) };

			_acc[0] += coeff0 * de_kernel_val[0];
			_acc[1] += coeff0 * de_kernel_val[1];
			_acc[2] += coeff0 * de_kernel_val[2];
		}
	}

	{
		acc_x += _acc[0];
		acc_y += _acc[1];
		acc_z += _acc[2];
	}

	// Body force
	{
		double volume_acc[3];

		if (CASE_DIM == 0) { volume_acc[0] = 0.0; volume_acc[1] = -GRAVITY; volume_acc[2] = 0.0; };
		if (CASE_DIM == 1) { volume_acc[0] = 0.0; volume_acc[1] = 0.0; volume_acc[2] = -GRAVITY; };

		acc_x += volume_acc[0];
		acc_y += volume_acc[1];
		acc_z += volume_acc[2];
	}

	/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

	{
		acc_x = constrain_d_gpu(acc_x, ACC_X_LIM_L, ACC_X_LIM_U);
		acc_y = constrain_d_gpu(acc_y, ACC_Y_LIM_L, ACC_Y_LIM_U);
		acc_z = constrain_d_gpu(acc_z, ACC_Z_LIM_L, ACC_Z_LIM_U);
	}

	/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

	particles[calcu_particle].accX.val[2] = particles[calcu_particle].accX.val[1];
	particles[calcu_particle].accX.val[1] = particles[calcu_particle].accX.val[0];
	particles[calcu_particle].accX.val[0] = acc_x;

	particles[calcu_particle].accY.val[2] = particles[calcu_particle].accY.val[1];
	particles[calcu_particle].accY.val[1] = particles[calcu_particle].accY.val[0];
	particles[calcu_particle].accY.val[0] = acc_y;

	particles[calcu_particle].accZ.val[2] = particles[calcu_particle].accZ.val[1];
	particles[calcu_particle].accZ.val[1] = particles[calcu_particle].accZ.val[0];
	particles[calcu_particle].accZ.val[0] = acc_z;
}

void acc_pressure_term_update_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * _neigh_list_cuda, PARTICLE * _particles_cuda, double dt, int blcks = 50)
{
	int offset;

	int _blcks = blcks; // how much GPUs launch at once
	int _thrds = 1;

	int cycle = (neigh_list_length / blcks) + 2;
	dim3 grid(_blcks, 1, 1);
	dim3 thrd(_thrds, 1, 1);

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_acc_pressure_term_update_schm1_cuda << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, offset);
		offset += blcks;
	}
}