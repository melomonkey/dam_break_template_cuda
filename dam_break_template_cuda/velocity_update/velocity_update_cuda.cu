#include "hip/hip_runtime.h"
#include "..\struct_particle.h"
#include "..\configuration.h"
#include "..\basic_func\basic_func.h"
#include <vector>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

__global__ void _velocity_update_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, double dt, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return; 

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	double h = particles[calcu_particle].smthR.val[0];

	// Velocity X update
	particles[calcu_particle].velX.val[2] = particles[calcu_particle].velX.val[1];
	particles[calcu_particle].velX.val[1] = particles[calcu_particle].velX.val[0];
	particles[calcu_particle].velX.val[0] += dt * particles[calcu_particle].accX.val[0];

	// Velocity Y update
	particles[calcu_particle].velY.val[2] = particles[calcu_particle].velY.val[1];
	particles[calcu_particle].velY.val[1] = particles[calcu_particle].velY.val[0];
	particles[calcu_particle].velY.val[0] += dt * particles[calcu_particle].accY.val[0];

	// Velocity Z update
	particles[calcu_particle].velZ.val[2] = particles[calcu_particle].velZ.val[1];
	particles[calcu_particle].velZ.val[1] = particles[calcu_particle].velZ.val[0];
	particles[calcu_particle].velZ.val[0] += dt * particles[calcu_particle].accZ.val[0];

	if (BOX_CONTAIN == 1)
	{
		if (particles[calcu_particle].coorX.val[0] <= BOX_X_MIN) particles[calcu_particle].velX.val[0] = 0.0;
		if (particles[calcu_particle].coorX.val[0] >= BOX_X_MAX) particles[calcu_particle].velX.val[0] = 0.0;

		if (particles[calcu_particle].coorY.val[0] <= BOX_Y_MIN) particles[calcu_particle].velY.val[0] = 0.0;
		if (particles[calcu_particle].coorY.val[0] >= BOX_Y_MAX) particles[calcu_particle].velY.val[0] = 0.0;

		if (particles[calcu_particle].coorZ.val[0] <= BOX_Z_MIN) particles[calcu_particle].velZ.val[0] = 0.0;
		if (particles[calcu_particle].coorZ.val[0] >= BOX_Z_MAX) particles[calcu_particle].velZ.val[0] = 0.0;

		if (CASE_DIM == 0) particles[calcu_particle].velZ.val[0] = 0.0;
	}

	{
		particles[calcu_particle].velX.val[0] = constrain_d_gpu(particles[calcu_particle].velX.val[0], VEL_X_LIM_L, VEL_X_LIM_U);
		particles[calcu_particle].velY.val[0] = constrain_d_gpu(particles[calcu_particle].velY.val[0], VEL_Y_LIM_L, VEL_Y_LIM_U);
		particles[calcu_particle].velZ.val[0] = constrain_d_gpu(particles[calcu_particle].velZ.val[0], VEL_Z_LIM_L, VEL_Z_LIM_U);
	}
}

void velocity_update_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * _neigh_list_cuda, PARTICLE * _particles_cuda, double dt, int blcks = 50)
{
	int offset;

	int _blcks = blcks; // how much GPUs launch at once
	int _thrds = 1;

	int cycle = (neigh_list_length / blcks) + 2;
	dim3 grid(_blcks, 1, 1);
	dim3 thrd(_thrds, 1, 1);

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_velocity_update_schm1_cuda << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, dt, offset);
		offset += blcks;
	}
}