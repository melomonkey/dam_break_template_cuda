#include "hip/hip_runtime.h"
#include "..\struct_particle.h"
#include "..\configuration.h"
#include "..\basic_func\basic_func.h"
#include <vector>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

__global__ void _velocity_filter_schm1_cuda_part1(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return;

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	double h = particles[calcu_particle].smthR.val[0];

	// The revisements of velocities in XYZ direction
	double tmp_velX_rev = 0.0,
		tmp_velY_rev = 0.0,
		tmp_velZ_rev = 0.0;

	double epsilon = 0.2;

	for (int j = 1; j < neigh_list_width; ++j)
	{
		if (neighbors_list[j + pos * neigh_list_width] == 0) break;

		int label_ij = neighbors_list[j + neigh_list_width * pos] - 1;

		double dis_x, dis_y, dis_z;
		dis_x = particles[label_ij].coorX.val[0] - particles[calcu_particle].coorX.val[0];
		dis_y = particles[label_ij].coorY.val[0] - particles[calcu_particle].coorY.val[0];
		dis_z = particles[label_ij].coorZ.val[0] - particles[calcu_particle].coorZ.val[0];

		double coeff0 = epsilon * particles[label_ij].mass.val[0];
		coeff0 /= (.5 * (particles[label_ij].density.val[0] + particles[calcu_particle].density.val[0]));

		double tmp_x, tmp_y, tmp_z;
		tmp_x = coeff0 * (particles[label_ij].velX.val[0] - particles[calcu_particle].velX.val[0]) * kernel_function_gpu(dis_x, h);
		tmp_y = coeff0 * (particles[label_ij].velY.val[0] - particles[calcu_particle].velY.val[0]) * kernel_function_gpu(dis_y, h);
		tmp_z = coeff0 * (particles[label_ij].velZ.val[0] - particles[calcu_particle].velZ.val[0]) * kernel_function_gpu(dis_z, h);

		tmp_velX_rev += tmp_x;
		tmp_velY_rev += tmp_y;
		tmp_velZ_rev += tmp_z;
	}

	particles[calcu_particle].velX.val[2] = particles[calcu_particle].velX.val[0] + tmp_velX_rev;
	particles[calcu_particle].velY.val[2] = particles[calcu_particle].velY.val[0] + tmp_velY_rev;
	particles[calcu_particle].velZ.val[2] = particles[calcu_particle].velZ.val[0] + tmp_velZ_rev;
}

__global__ void _velocity_filter_schm1_cuda_part2(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return;

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	particles[calcu_particle].velX.val[0] = particles[calcu_particle].velX.val[2];
	particles[calcu_particle].velY.val[0] = particles[calcu_particle].velY.val[2];
	particles[calcu_particle].velZ.val[0] = particles[calcu_particle].velZ.val[2];
}

void velocity_filter_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * _neigh_list_cuda, PARTICLE * _particles_cuda, int blcks = 500)
{
	int offset;

	int _blcks = blcks; // how much GPUs launch at once
	int _thrds = 1;

	int cycle = (neigh_list_length / blcks) + 2;
	dim3 grid(_blcks, 1, 1);
	dim3 thrd(_thrds, 1, 1);

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_velocity_filter_schm1_cuda_part1 << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, offset);
		offset += blcks;
	}

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_velocity_filter_schm1_cuda_part2 << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, offset);
		offset += blcks;
	}
}