#include "hip/hip_runtime.h"
#include "..\struct_particle.h"
#include "..\configuration.h"
#include "..\basic_func\basic_func.h"
#include <vector>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

__global__ void _pressure_update_schm1(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset; 

	if (pos > neigh_list_length - 1) return; 

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	double density_ratio = particles[calcu_particle].density.val[0] / REF_DENSITY;
	particles[calcu_particle].pressure.val[0] = COMPRS_LIM_CONST_B * (pow((density_ratio), 7.0) - 1.0);

	particles[calcu_particle].pressure.val[0] = constrain_d_gpu(particles[calcu_particle].pressure.val[0], PRESSURE_LIM_L, PRESSURE_LIM_U);
}

void pressure_update_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * _neigh_list_cuda, PARTICLE * _particles_cuda, int blcks = 50)
{
	int offset;

	int _blcks = blcks; // how much GPUs launch at once
	int _thrds = 1;

	int cycle = (neigh_list_length / blcks) + 2;
	dim3 grid(_blcks, 1, 1);
	dim3 thrd(_thrds, 1, 1);

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_pressure_update_schm1 << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, offset);
		offset += blcks;
	}
}