#include "hip/hip_runtime.h"
#include "..\struct_particle.h"
#include "..\configuration.h"
#include "..\basic_func\basic_func.h"
#include <vector>
using namespace std;

#include "hip/hip_runtime.h"
#include ""


__global__ void _density_filter_schm1_cuda_part1(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return;

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	double local_x, local_y, local_z;
	local_x = particles[calcu_particle].coorX.val[0];
	local_y = particles[calcu_particle].coorY.val[0];
	local_z = particles[calcu_particle].coorZ.val[0];
	double h = particles[calcu_particle].smthR.val[0];

	double _sum1 = 0.0;
	double _sum2 = 0.0;

	for (int j = 0; j < neigh_list_width; ++j)
	{
		if (neighbors_list[j + pos * neigh_list_width] == 0) break;

		unsigned int label_ij = neighbors_list[j + pos * neigh_list_width] - 1;

		if (label_ij < 0) break;

		if (particles[label_ij].id == 0 || particles[label_ij].id == 1 || particles[label_ij].id == 2 || particles[label_ij].id == 3) continue;

		double dis_x, dis_y, dis_z;
		dis_x = local_x - particles[label_ij].coorX.val[0];
		dis_y = local_y - particles[label_ij].coorY.val[0];
		dis_z = local_z - particles[label_ij].coorZ.val[0];

		double kernel_val[3] = { kernel_function_gpu(dis_x, h), kernel_function_gpu(dis_y, h), kernel_function_gpu(dis_z, h) };


		if (particles[label_ij].density.val[0] == 0.0) continue;

		double tmp_volume = particles[label_ij].mass.val[0] / particles[label_ij].density.val[0];

		_sum1 += (particles[label_ij].density.val[0]) * tmp_volume * kernel_val[0];
		_sum1 += (particles[label_ij].density.val[0]) * tmp_volume * kernel_val[1];
		if (CASE_DIM == 1) _sum1 += (particles[label_ij].density.val[0]) * tmp_volume * kernel_val[2];

		_sum2 += tmp_volume * kernel_val[0];
		_sum2 += tmp_volume * kernel_val[1];
		if (CASE_DIM == 1) _sum2 += tmp_volume * kernel_val[2];
	}

	if (_sum2 == 0.0) return;

	particles[calcu_particle].density.val[2] = _sum1 / _sum2;
}

__global__ void _density_filter_schm1_cuda_part2(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return;

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	particles[calcu_particle].density.val[0] = particles[calcu_particle].density.val[2];
}

void density_filter_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * _neigh_list_cuda, PARTICLE * _particles_cuda, int blcks = 500)
{
	int offset;

	int _blcks = blcks; // how much GPUs launch at once
	int _thrds = 1;

	int cycle = (neigh_list_length / blcks) + 2;
	dim3 grid(_blcks, 1, 1);
	dim3 thrd(_thrds, 1, 1);

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_density_filter_schm1_cuda_part1 << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, offset);
		offset += blcks;
	}

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_density_filter_schm1_cuda_part2 << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, offset);
		offset += blcks;
	}
}