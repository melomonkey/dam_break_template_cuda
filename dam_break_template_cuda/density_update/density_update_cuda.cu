#include "hip/hip_runtime.h"
#include "..\struct_particle.h"
#include "..\configuration.h"
#include "..\basic_func\basic_func.h"
#include <vector>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

__global__ void _density_update_schm1_cuda_part1(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return;

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

    double local_x, local_y, local_z;
	local_x = particles[calcu_particle].coorX.val[0];
	local_y = particles[calcu_particle].coorY.val[0];
	local_z = particles[calcu_particle].coorZ.val[0];
	double h = particles[calcu_particle].smthR.val[0];

	double tmp_dev = 0.0;
	for (int j = 1; j < neigh_list_width; ++j)
	{
		if (neighbors_list[j + pos * neigh_list_width] == 0) break;

		unsigned int label_ij = neighbors_list[j + pos * neigh_list_width] - 1;

		if (particles[label_ij].id == 0 || particles[label_ij].id == 1 || particles[label_ij].id == 2 || particles[label_ij].id == 3) continue;

		double tmp_x, tmp_y, tmp_z;
		tmp_x = particles[label_ij].coorX.val[0];
		tmp_y = particles[label_ij].coorY.val[0];
		tmp_z = particles[label_ij].coorZ.val[0];

		double dis_x, dis_y, dis_z;
		dis_x = local_x - tmp_x;
		dis_y = local_y - tmp_y;
		dis_z = local_z - tmp_z;

		double coeff0 = particles[calcu_particle].density.val[0] * particles[label_ij].mass.val[0] / particles[label_ij].density.val[0];

		tmp_dev += coeff0 * (particles[calcu_particle].velX.val[0] - particles[label_ij].velX.val[0]) * kernel_function_1dev_gpu(dis_x, h);
		tmp_dev += coeff0 * (particles[calcu_particle].velY.val[0] - particles[label_ij].velY.val[0]) * kernel_function_1dev_gpu(dis_y, h);
		if (CASE_DIM == 1) tmp_dev += coeff0 * (particles[calcu_particle].velZ.val[0] - particles[label_ij].velZ.val[0]) * kernel_function_1dev_gpu(dis_z, h);
	}

	double tmp = tmp_dev;

	particles[calcu_particle].density.val[1] = tmp;
}

__global__ void _density_update_schm1_cuda_part2(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, double dt, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return;

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	double change = particles[calcu_particle].density.val[1] * dt;
	change = constrain_d_gpu(change, -DENS_CHANGE_LIM, DENS_CHANGE_LIM);

	particles[calcu_particle].density.val[0] += change;
	particles[calcu_particle].density.val[0] = constrain_d_gpu(particles[calcu_particle].density.val[0], DENS_LIM_L, DENS_LIM_U);
}

void density_update_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * _neigh_list_cuda, PARTICLE * _particles_cuda, double dt, int blcks = 50)
{
	int offset;

	int _blcks = blcks; // how much GPUs launch at once
	int _thrds = 1;

	int cycle = (neigh_list_length / blcks) + 2;
	dim3 grid(_blcks, 1, 1);
	dim3 thrd(_thrds, 1, 1);

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_density_update_schm1_cuda_part1 << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, offset);
		offset += blcks;
	}

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_density_update_schm1_cuda_part2 << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, dt, offset);
		offset += blcks;
	}
}