#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "struct_particle.h"
#include <stdio.h>
#include "global_variables.h"

void _neighbors_list_IN_gpu()
{
	unsigned int mx_idx = neighbors_list[0].size();

	for (int i = 0; i < neighbors_list.size(); ++i)
	{
		if (neighbors_list[i].size() > mx_idx) mx_idx = neighbors_list[i].size();
	}

	_neigh_list_row = neighbors_list.size();
	_neigh_list_col = mx_idx;

	_neighbors_list_buff = new unsigned int[_neigh_list_row * _neigh_list_col]; 
	for (int i = 0; i < _neigh_list_row * _neigh_list_col; ++i) _neighbors_list_buff[i] = 0; 

	for (int i = 0; i < neighbors_list.size(); ++i)
	{
		for (int j = 0; j < neighbors_list[i].size(); ++j)
		{
			_neighbors_list_buff[j + i * _neigh_list_col] = neighbors_list[i][j]; 
		}
	}

	hipFree(_neighbors_list_cuda);
	cudaStatus = hipMalloc((void **)&_neighbors_list_cuda, sizeof(unsigned int) * _neigh_list_row * _neigh_list_col);
	cudaStatus = hipMemcpy(_neighbors_list_cuda, _neighbors_list_buff, sizeof(unsigned int) * _neigh_list_row * _neigh_list_col, hipMemcpyHostToDevice);
	
	free(_neighbors_list_buff); 
}

void _coordination_list_OUT_gpu()
{
	
	_coordination_buff = new double[3 * _size_particles];

	cudaStatus = hipMemcpy(_coordination_buff, _coordination_cuda, sizeof(double) * 3 * _size_particles, hipMemcpyDeviceToHost);

	coordination.clear(); 
	coordination.reserve(particles.size()); 
	for (int i = 0; i < particles.size(); ++i)
	{
		vector<double> tmp(3, 0.0);
		tmp[0] = _coordination_buff[0 + i * 3];
		tmp[1] = _coordination_buff[1 + i * 3];
		tmp[2] = _coordination_buff[2 + i * 3];

		coordination.push_back(tmp);
	}

	free(_coordination_buff);
}

void _initialize_lists()
{
	_size_particles = particles.size(); 

	_particles_buff = new PARTICLE[_size_particles]; 

	cudaStatus = hipMalloc((void **)&_particles_cuda, sizeof(PARTICLE) * _size_particles);

	cudaStatus = hipMalloc((void **)&_coordination_cuda, sizeof(double) * 3 * _size_particles);

	cudaStatus = hipMalloc((void **)&_neighbors_list_cuda, sizeof(unsigned int) * 2);

	coordination.clear();
	coordination.reserve(particles.size());
	for (int i = 0; i < particles.size(); ++i)
	{
		vector<double> tmp(3, 0.0);
		tmp[0] = particles[i].coorX.val[0];
		tmp[1] = particles[i].coorY.val[0];
		tmp[2] = particles[i].coorZ.val[0];

		coordination.push_back(tmp);
	}
}

void _particles_IN_gpu()
{
	for (int i = 0; i < particles.size(); ++i) _particles_buff[i] = particles[i];

	cudaStatus = hipMalloc((void **)&_particles_cuda, sizeof(PARTICLE) * _size_particles);
	cudaStatus = hipMemcpy(_particles_cuda, _particles_buff, sizeof(PARTICLE) * _size_particles, hipMemcpyHostToDevice);

	
	/*__global__ void update_coordination_list_gpu(unsigned int _size, double * _coordination_list, PARTICLE * _particles);
	dim3 grid(1, 1, 1);
	dim3 thrd(1, 1, 1);

	update_coordination_list_gpu <<<grid, thrd>>> (_size_particles, _coordination_cuda, _particles_cuda);*/
}

void particles_OUT_gpu()
{
	cudaStatus = hipMemcpy(_particles_buff, _particles_cuda, sizeof(PARTICLE) * _size_particles, hipMemcpyDeviceToHost);

	for (int i = 0; i < particles.size(); ++i) particles[i] = _particles_buff[i]; 
}

__global__ void _update_coordination_list_gpu(unsigned int _size, double * _coordination_list_cuda, PARTICLE * _particles)
{
	for (int i = 0; i < _size; ++i)
	{
		_coordination_list_cuda[0 + i * 3] = _particles[i].coorX.val[0];
		_coordination_list_cuda[1 + i * 3] = _particles[i].coorY.val[0];
		_coordination_list_cuda[2 + i * 3] = _particles[i].coorZ.val[0];
	}
}

void update_coordination_list_gpu()
{
	dim3 grid(1, 1, 1);
	dim3 thrd(1, 1, 1);

	_update_coordination_list_gpu << <grid, thrd >> > (_size_particles, _coordination_cuda, _particles_cuda);
}