#include "hip/hip_runtime.h"
#include "..\struct_particle.h"
#include "..\configuration.h"
#include "..\basic_func\basic_func.h"
#include <vector>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

__global__ void _postion_time_update_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, double dt, unsigned int offset)
{
	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return;

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	if (particles[calcu_particle].id == 0 || particles[calcu_particle].id == 1 || particles[calcu_particle].id == 2 || particles[calcu_particle].id == 3) return;

	// Coordination X update
	particles[calcu_particle].coorX.val[2] = particles[calcu_particle].coorX.val[1];
	particles[calcu_particle].coorX.val[1] = particles[calcu_particle].coorX.val[0];
	particles[calcu_particle].coorX.val[0] += dt * particles[calcu_particle].velX.val[0];


	// Coordination Y update
	particles[calcu_particle].coorY.val[2] = particles[calcu_particle].coorY.val[1];
	particles[calcu_particle].coorY.val[1] = particles[calcu_particle].coorY.val[0];
	particles[calcu_particle].coorY.val[0] += dt * particles[calcu_particle].velY.val[0];


	// Coordination Z update
	particles[calcu_particle].coorZ.val[2] = particles[calcu_particle].coorZ.val[1];
	particles[calcu_particle].coorZ.val[1] = particles[calcu_particle].coorZ.val[0];
	particles[calcu_particle].coorZ.val[0] += dt * particles[calcu_particle].velZ.val[0];

	// * Box constrain
	if (BOX_CONTAIN == 1)
	{
		if (particles[calcu_particle].coorX.val[0] < BOX_X_MIN) particles[calcu_particle].coorX.val[0] = BOX_X_MIN;
		if (particles[calcu_particle].coorX.val[0] > BOX_X_MAX) particles[calcu_particle].coorX.val[0] = BOX_X_MAX;

		if (particles[calcu_particle].coorY.val[0] < BOX_Y_MIN) particles[calcu_particle].coorY.val[0] = BOX_Y_MIN;
		if (particles[calcu_particle].coorY.val[0] > BOX_Y_MAX) particles[calcu_particle].coorY.val[0] = BOX_Y_MAX;

		if (particles[calcu_particle].coorZ.val[0] < BOX_Z_MIN) particles[calcu_particle].coorZ.val[0] = BOX_Z_MIN;
		if (particles[calcu_particle].coorZ.val[0] > BOX_Z_MAX) particles[calcu_particle].coorZ.val[0] = BOX_Z_MAX;

		if (CASE_DIM == 0) particles[calcu_particle].coorZ.val[0] = 0.0;
	}

	particles[calcu_particle].time.val[2] = particles[calcu_particle].time.val[1];
	particles[calcu_particle].time.val[1] = particles[calcu_particle].time.val[0];
	particles[calcu_particle].time.val[0] += dt;
}

void position_time_update_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * _neigh_list_cuda, PARTICLE * _particles_cuda, double dt, int blcks = 50)
{
	int offset;

	int _blcks = blcks; // how much GPUs launch at once
	int _thrds = 1;

	int cycle = (neigh_list_length / blcks) + 2;
	dim3 grid(_blcks, 1, 1);
	dim3 thrd(_thrds, 1, 1);

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_postion_time_update_cuda << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, dt, offset);
		offset += blcks;
	}
}