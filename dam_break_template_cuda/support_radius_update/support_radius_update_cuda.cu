#include "hip/hip_runtime.h"
#include "..\struct_particle.h"
#include "..\configuration.h"
#include "..\basic_func\basic_func.h"
#include <vector>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

__global__ void _support_radius_update_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * neighbors_list, PARTICLE * particles, double dt, unsigned int offset)
{
	double dimension;
	if (CASE_DIM == 0) dimension = 2.0;
	if (CASE_DIM == 1) dimension = 3.0;

	unsigned int pos = blockIdx.x + offset;

	if (pos > neigh_list_length - 1) return; 

	unsigned int calcu_particle = neighbors_list[0 + neigh_list_width * pos] - 1;

	double density_dev;
	density_dev = particles[calcu_particle].density.val[1];

	double tmp_h = -(1.0 / dimension) * (particles[calcu_particle].smthR.val[0] / particles[calcu_particle].density.val[0]) * density_dev;

	particles[calcu_particle].smthR.val[2] = particles[calcu_particle].smthR.val[1];
	particles[calcu_particle].smthR.val[1] = particles[calcu_particle].smthR.val[0];
	particles[calcu_particle].smthR.val[0] += tmp_h * dt;
}

void support_radius_update_schm1_cuda(unsigned int neigh_list_length, unsigned int neigh_list_width, unsigned int * _neigh_list_cuda, PARTICLE * _particles_cuda, double dt, int blcks = 500)
{
	int offset;

	int _blcks = blcks; // how much GPUs launch at once
	int _thrds = 1;

	int cycle = (neigh_list_length / blcks) + 2;
	dim3 grid(_blcks, 1, 1);
	dim3 thrd(_thrds, 1, 1);

	offset = 0;
	for (int i = 0; i < cycle; ++i)
	{
		_support_radius_update_schm1_cuda << <grid, thrd >> >(neigh_list_length, neigh_list_width, _neigh_list_cuda, _particles_cuda, dt, offset);
		offset += blcks;
	}
}